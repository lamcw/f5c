#include <errno.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_fp16.h>

#include "error.h"
#include "f5c.h"
#include "f5cmisc.cuh"
#include "f5cmisc.h"

void init_cuda(core_t* core){

    core->cuda = (cuda_data_t*)malloc(sizeof(cuda_data_t)); 
    MALLOC_CHK(core->cuda);

    core->align_kernel_time=0;
    core->align_pre_kernel_time=0;
    core->align_core_kernel_time=0;
    core->align_post_kernel_time=0;
    core->align_cuda_malloc=0;
    core->extra_load_cpu=0;
    core->align_cuda_memcpy=0;
    core->align_cuda_postprocess=0;
    core->align_cuda_preprocess=0;

#ifdef CUDA_PRE_MALLOC

    int32_t n_bam_rec = core->opt.batch_size;
    //cpu arrays
    core->cuda->read_ptr_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(core->cuda->read_ptr_host);
    core->cuda->n_events_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(core->cuda->n_events_host);
    core->cuda->event_ptr_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(core->cuda->event_ptr_host);

    core->cuda->read_len_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(core->cuda->read_len_host);
    core->cuda->scalings_host = (scalings_t*)malloc(sizeof(scalings_t) * n_bam_rec);
    MALLOC_CHK(core->cuda->scalings_host);
    core->cuda->n_event_align_pairs_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(core->cuda->n_event_align_pairs_host);

    //cuda arrays
    print_size("read_ptr array",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&(core->cuda->read_ptr), n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    print_size("read_lens",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&(core->cuda->read_len), n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //n_events
    print_size("n_events",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&(core->cuda->n_events), n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //event ptr
    print_size("event ptr",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&(core->cuda->event_ptr), n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //scalings : already linear
    print_size("Scalings",n_bam_rec * sizeof(scalings_t));
    hipMalloc((void**)&(core->cuda->scalings), n_bam_rec * sizeof(scalings_t));
    CUDA_CHK();
    hipMalloc((void**)&(core->cuda->model),
            NUM_KMER * sizeof(model_t));
    CUDA_CHK();  

    print_size("n_event_align_pairs",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&(core->cuda->n_event_align_pairs), n_bam_rec * sizeof(int32_t));
    CUDA_CHK();

    //model : already linear //move to cuda_init
    hipMemcpy(core->cuda->model, core->model, NUM_KMER * sizeof(model_t),
    hipMemcpyHostToDevice);
    CUDA_CHK();

    

#endif

    return;
}

void free_cuda(core_t* core){

#ifdef CUDA_PRE_MALLOC
    free(core->cuda->event_ptr_host);
    free(core->cuda->n_events_host);
    free(core->cuda->read_ptr_host);
    free(core->cuda->read_len_host);
    free(core->cuda->scalings_host);
    free(core->cuda->n_event_align_pairs_host);
    
    hipFree(core->cuda->read_ptr);
    hipFree(core->cuda->read_len);
    hipFree(core->cuda->n_events);
    hipFree(core->cuda->event_ptr);
    hipFree(core->cuda->model); //constant memory
    hipFree(core->cuda->scalings);
    hipFree(core->cuda->n_event_align_pairs);

#endif

    free(core->cuda);
    return;
}


#ifndef CPU_GPU_PROC

void align_cuda(core_t* core, db_t* db) {
    int32_t i;
    int32_t n_bam_rec = db->n_bam_rec;
    double realtime1;

    /**cuda pointers*/
    char* read;        //flattened reads sequences
    int32_t* read_ptr; //index pointer for flattedned "reads"
    int32_t* read_len;
    int64_t sum_read_len;
    int32_t* n_events;
    event_t* event_table;
    int32_t* event_ptr;
    int64_t sum_n_events;
    scalings_t* scalings;
    AlignedPair* event_align_pairs;
    int32_t* n_event_align_pairs;
    float *bands;
    uint8_t *trace;
    EventKmerPair* band_lower_left;

realtime1 = realtime();

#ifdef CUDA_PRE_MALLOC
    int32_t* read_ptr_host = core->cuda->read_ptr_host;
#else
    //get the total size and create the pointers
    int32_t* read_ptr_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(read_ptr_host);
#endif
    sum_read_len = 0;

    //read sequences : needflattening
    for (i = 0; i < n_bam_rec; i++) {
        read_ptr_host[i] = sum_read_len;
        sum_read_len += (db->read_len[i] + 1); //with null term
    }
    //form the temporary flattened array on host
    char* read_host = (char*)malloc(sizeof(char) * sum_read_len);
    MALLOC_CHK(read_host);
    for (i = 0; i < n_bam_rec; i++) {
        int32_t idx = read_ptr_host[i];
        strcpy(&read_host[idx], db->read[i]);
    }

    //now the events : need flattening
    //num events : need flattening
    //get the total size and create the pointers
#ifdef CUDA_PRE_MALLOC
    int32_t* n_events_host = core->cuda->n_events_host;
    int32_t* event_ptr_host = core->cuda->event_ptr_host;
#else
    int32_t* n_events_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(n_events_host);
    int32_t* event_ptr_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(event_ptr_host);
#endif

    sum_n_events = 0;
    for (i = 0; i < n_bam_rec; i++) {
        n_events_host[i] = db->et[i].n;
        event_ptr_host[i] = sum_n_events;
        sum_n_events += db->et[i].n;
    }

    //event table flatten
    //form the temporary flattened array on host
    event_t* event_table_host =
        (event_t*)malloc(sizeof(event_t) * sum_n_events);
    MALLOC_CHK(event_table_host);
    for (i = 0; i < n_bam_rec; i++) {
        int32_t idx = event_ptr_host[i];
        memcpy(&event_table_host[idx], db->et[i].event,
               sizeof(event_t) * db->et[i].n);
    }

    AlignedPair* event_align_pairs_host =
        (AlignedPair*)malloc(2 * sum_n_events * sizeof(AlignedPair));
    MALLOC_CHK(event_align_pairs_host);

core->align_cuda_preprocess += (realtime() - realtime1);

    /** Start GPU mallocs**/
realtime1 = realtime();

#ifdef CUDA_PRE_MALLOC
    read_ptr =core->cuda->read_ptr;
    read_len=core->cuda->read_len;
    n_events=core->cuda->n_events;
    event_ptr=core->cuda->event_ptr;
    scalings=core->cuda->scalings;
    model_t* model = core->cuda->model;
#else

    print_size("read_ptr array",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&read_ptr, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();

    print_size("read_lens",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&read_len, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //n_events
    print_size("n_events",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&n_events, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //event ptr
    print_size("event ptr",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&event_ptr, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    //scalings : already linear
    print_size("Scalings",n_bam_rec * sizeof(scalings_t));
    hipMalloc((void**)&scalings, n_bam_rec * sizeof(scalings_t));
    CUDA_CHK();
    //model : already linear
    model_t* model;
    hipMalloc((void**)&model,
            NUM_KMER * sizeof(model_t));
    CUDA_CHK();  
#endif


    print_size("read array",sum_read_len * sizeof(char));
    hipMalloc((void**)&read, sum_read_len * sizeof(char)); //with null char
    CUDA_CHK();
    print_size("event table",sum_n_events * sizeof(event_t));
    hipMalloc((void**)&event_table, sum_n_events * sizeof(event_t));
    CUDA_CHK();
    model_t* model_kmer_cache;
    hipMalloc((void**)&model_kmer_cache, sum_read_len * sizeof(model_t)); 
    CUDA_CHK();
 
    /**allocate output arrays for cuda**/
    print_size("event align pairs",2 * sum_n_events *sizeof(AlignedPair));
    hipMalloc((void**)&event_align_pairs,
            2 * sum_n_events *
                sizeof(AlignedPair)); //todo : need better huristic
    CUDA_CHK();
#ifdef CUDA_PRE_MALLOC
    n_event_align_pairs=core->cuda->n_event_align_pairs;
#else
    print_size("n_event_align_pairs",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&n_event_align_pairs, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
#endif
    //scratch arrays
    size_t sum_n_bands = sum_n_events + sum_read_len; //todo : can be optimised 
    print_size("bands",sizeof(float) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&bands,sizeof(float) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    print_size("trace",sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&trace, sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    hipMemset(trace,0,sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH); //initialise the trace array to 0
    print_size("band_lower_left",sizeof(EventKmerPair)* sum_n_bands);
    hipMalloc((void**)&band_lower_left, sizeof(EventKmerPair)* sum_n_bands);
    CUDA_CHK();   
core->align_cuda_malloc += (realtime() - realtime1);

    /* cuda mem copys*/
realtime1 =realtime();
    hipMemcpy(read_ptr, read_ptr_host, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(read, read_host, sum_read_len * sizeof(char),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    //read length : already linear hence direct copy
    hipMemcpy(read_len, db->read_len, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(n_events, n_events_host, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(event_ptr, event_ptr_host, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(event_table, event_table_host, sizeof(event_t) * sum_n_events,
               hipMemcpyHostToDevice);
    CUDA_CHK();

#ifndef CUDA_PRE_MALLOC
//model : already linear //move to cuda_init
    hipMemcpy(model, core->model, NUM_KMER * sizeof(model_t),
            hipMemcpyHostToDevice);
    CUDA_CHK();
#endif
    //can be interleaved
    hipMemcpy(scalings, db->scalings, sizeof(scalings_t) * n_bam_rec,
               hipMemcpyHostToDevice);
    CUDA_CHK();
core->align_cuda_memcpy += (realtime() - realtime1);



realtime1 = realtime();    
 
    /*pre kernel*/
    assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
    dim3 gridpre(1,(db->n_bam_rec + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
    dim3 blockpre(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);  
	fprintf(stderr,"grid %d,%d, block %d,%d\n",gridpre.x,gridpre.y, blockpre.x,blockpre.y);	

    align_kernel_pre_2d<<<gridpre, blockpre>>>( read,
        read_len, read_ptr, n_events,
        event_ptr, model, n_bam_rec, model_kmer_cache,bands,trace,band_lower_left); 
       
    hipDeviceSynchronize();CUDA_CHK();
    fprintf(stderr, "[%s::%.3f*%.2f] align pre done\n", __func__,
            realtime() - realtime1, cputime() / (realtime() - realtime1));
core->align_kernel_time += (realtime() - realtime1);        
core->align_pre_kernel_time += (realtime() - realtime1);        
                
realtime1 = realtime();

    /* core kernel*/
    assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
    dim3 grid1(1,(db->n_bam_rec + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
    dim3 block1(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);
    align_kernel_core_2d_shm<<<grid1, block1>>>(read_len, read_ptr, event_table, n_events,
            event_ptr, scalings, n_bam_rec, model_kmer_cache,bands,trace,band_lower_left );
    
    hipDeviceSynchronize();CUDA_CHK();
    fprintf(stderr, "[%s::%.3f*%.2f] align done\n", __func__,
    realtime() - realtime1, cputime() / (realtime() - realtime1));
    core->align_kernel_time += (realtime() - realtime1);
core->align_core_kernel_time += (realtime() - realtime1);

realtime1 = realtime();
    
    /*post kernel*/
    int32_t BLOCK_LEN = core->opt.cuda_block_size;
    dim3 gridpost((db->n_bam_rec + BLOCK_LEN - 1) / BLOCK_LEN);
    dim3 blockpost(BLOCK_LEN);
    #ifndef WARP_HACK  
        align_kernel_post<<<gridpost, blockpost>>>(event_align_pairs, n_event_align_pairs,
            read_len, read_ptr, event_table, n_events,
            event_ptr,scalings, n_bam_rec, model_kmer_cache,bands,trace,band_lower_left );     

    #else
        assert(BLOCK_LEN>=32);    
        dim3 grid1post((db->n_bam_rec + (BLOCK_LEN/32) - 1) / (BLOCK_LEN/32)); 
        fprintf(stderr,"grid new %d\n",grid1post.x);   
        align_kernel_post<<<grid1post, blockpost>>>(event_align_pairs, n_event_align_pairs,
            read_len, read_ptr, event_table, n_events,
            event_ptr, scalings, n_bam_rec, model_kmer_cache,bands,trace,band_lower_left );
    #endif
    hipDeviceSynchronize();CUDA_CHK();
    fprintf(stderr, "[%s::%.3f*%.2f] align post done\n", __func__,
            realtime() - realtime1, cputime() / (realtime() - realtime1));
    core->align_kernel_time += (realtime() - realtime1);        
core->align_post_kernel_time += (realtime() - realtime1);        


    //fprintf(stderr,"readlen %d,n_events %d\n",db->read_len[i],n_event_align_pairs);

#ifdef CUDA_DEBUG

    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    //todo : print a message to detect the launch timed out
    if (code == hipErrorLaunchTimeOut) {
        ERROR("%s", "The kernel timed out. You have to first disable the cuda "
                    "time out.");
        fprintf(
            stderr,
            "On Ubuntu do the following\nOpen the file /etc/X11/xorg.conf\nYou "
            "will have a section about your NVIDIA device. Add the following "
            "line to it.\nOption \"Interactive\" \"0\"\nIf you do not have a "
            "section about your NVIDIA device in /etc/X11/xorg.conf or you do "
            "not have a file named /etc/X11/xorg.conf, run the command sudo "
            "nvidia-xconfig to generate a xorg.conf file and do as above.\n\n");
    }
    if (code != hipSuccess) {
        fprintf(stderr, "Cuda error: %s \n in file : %s line number : %lu\n",
                hipGetErrorString(code), __FILE__, __LINE__);
        exit(-1);
    }        
    
#endif

    /** copyback ans**/
realtime1 =  realtime();
    hipMemcpy(db->n_event_align_pairs, n_event_align_pairs,
               n_bam_rec * sizeof(int32_t), hipMemcpyDeviceToHost);
    CUDA_CHK();
    
    hipMemcpy(event_align_pairs_host, event_align_pairs,
               2 * sum_n_events * sizeof(AlignedPair), hipMemcpyDeviceToHost);
    CUDA_CHK();
core->align_cuda_memcpy += (realtime() - realtime1);

realtime1 =  realtime();
#ifndef CUDA_PRE_MALLOC
    hipFree(read_ptr);
    hipFree(read_len);
    hipFree(n_events);
    hipFree(event_ptr);
    hipFree(model); //constant memory
    hipFree(scalings);
    hipFree(n_event_align_pairs);
#endif
    hipFree(read); //with null char
    hipFree(event_table);
    hipFree(event_align_pairs);
    hipFree(bands);
    hipFree(trace);
    hipFree(band_lower_left);
    hipFree(model_kmer_cache);

core->align_cuda_malloc += (realtime() - realtime1);    
    
    /** post work**/
realtime1 =  realtime();
    //copy back
    for (i = 0; i < n_bam_rec; i++) {
        int32_t idx = event_ptr_host[i];
        memcpy(db->event_align_pairs[i], &event_align_pairs_host[idx * 2],
               sizeof(AlignedPair) * db->n_event_align_pairs[i]);
    }

    //free the temp arrays on host
#ifndef CUDA_PRE_MALLOC
    free(read_ptr_host);
    free(n_events_host);
    free(event_ptr_host);
#endif
    free(read_host);
    free(event_table_host);
    free(event_align_pairs_host);


core->align_cuda_postprocess += (realtime() - realtime1);

}



#else



#ifdef WORK_STEAL
static inline int32_t steal_work(pthread_arg_t* all_args, int32_t n_threads)
{

	int32_t i, c_i = -1;
	int32_t k;
	for (i = 0; i < n_threads; ++i){
        pthread_arg_t args = all_args[i];
        //fprintf(stderr,"endi : %d, starti : %d\n",args.endi,args.starti);
		if (args.endi-args.starti > STEAL_THRESH) {
            //fprintf(stderr,"gap : %d\n",args.endi-args.starti);
            c_i = i;
            break;
        }
    }
    if(c_i<0){
        return -1;
    }
	k = __sync_fetch_and_add(&(all_args[c_i].starti), 1);
    //fprintf(stderr,"k : %d, end %d, start %d\n",k,all_args[c_i].endi,all_args[c_i].starti);
	return k >= all_args[c_i].endi ? -1 : k;
}
#endif

void* pthread_cusingle(void* voidargs) {
    int32_t i,j;
    pthread_arg_t* args = (pthread_arg_t*)voidargs;
    db_t* db = args->db;
    core_t* core = args->core;

#ifndef WORK_STEAL
    for (i = args->starti; i < args->endi; i++) {
        j=args->ultra_long_reads[i];
        args->func(core,db,j);
    }
#else
    pthread_arg_t* all_args = (pthread_arg_t*)(args->all_pthread_args);
    //adapted from ktherad
	for (;;) {
		i = __sync_fetch_and_add(&args->starti, 1);
		if (i >= args->endi) {
            break;
        }
        j=args->ultra_long_reads[i];
		args->func(core,db,j);
	}
	while ((i = steal_work(all_args,core->opt.num_thread)) >= 0){
        j=args->ultra_long_reads[i];
		args->func(core,db,j);  
    }  
#endif

    //fprintf(stderr,"Thread %d done\n",(myargs->position)/THREADS);
    pthread_exit(0);
}


void pthread_cudb(core_t* core, db_t* db, int32_t* ultra_long_reads, int32_t  n_ultra_long_reads,void (*func)(core_t*,db_t*,int)){ 
    //create threads
    pthread_t tids[core->opt.num_thread];
    pthread_arg_t pt_args[core->opt.num_thread];
    int32_t t, ret;
    int32_t i = 0;
    int32_t num_thread = core->opt.num_thread;
    int32_t step = (n_ultra_long_reads + num_thread - 1) / num_thread;
    //todo : check for higher num of threads than the data
    //current works but many threads are created despite 

    //set the data structures
    for (t = 0; t < num_thread; t++) {
        pt_args[t].core = core;
        pt_args[t].db = db;
        pt_args[t].starti = i;
        i += step;
        if (i > n_ultra_long_reads) {
            pt_args[t].endi = n_ultra_long_reads;
        } else {
            pt_args[t].endi = i;
        }
        pt_args[t].func=func;
        pt_args[t].ultra_long_reads=ultra_long_reads;
    #ifdef WORK_STEAL    
        pt_args[t].all_pthread_args =  (void *)pt_args;
    #endif
        //fprintf(stderr,"t%d : %d-%d\n",t,pt_args[t].starti,pt_args[t].endi);

    }

    //create threads
    for(t = 0; t < core->opt.num_thread; t++){
        ret = pthread_create(&tids[t], NULL, pthread_cusingle,
                                (void*)(&pt_args[t]));
        NEG_CHK(ret);
    }

    //pthread joining
    for (t = 0; t < core->opt.num_thread; t++) {
        int ret = pthread_join(tids[t], NULL);
        NEG_CHK(ret);
    }
}


void* align_cudb(void* voidargs){

    pthread_arg_t* args = (pthread_arg_t*)voidargs;
    db_t* db = args->db;
    core_t* core = args->core;
    int32_t* ultra_long_reads = args->ultra_long_reads;
    int32_t n_ultra_long_reads = args->endi;
    //fprintf(stderr,"ultra long guys : %d\n",n_ultra_long_reads);
    //fprintf(stderr, "cpu\n");
    if (core->opt.num_thread == 1) {
        int j;
        for(j=0;j<n_ultra_long_reads;j++) {
            int32_t i = ultra_long_reads[j];
            align_single(core, db, i);
            // db->n_event_align_pairs[i] =
            //     align(db->event_align_pairs[i], db->read[i],
            //           db->read_len[i], db->et[i], core->model,
            //           db->scalings[i], db->f5[i]->sample_rate);
            //fprintf(stderr,"readlen %d,n_events %d\n",db->read_len[i],n_event_align_pairs);
        }
    } else {
        pthread_cudb(core, db, ultra_long_reads,n_ultra_long_reads,align_single);
        
    }

    fprintf(stderr,"%d reads (length>%d kb) processed on cpu\n",n_ultra_long_reads,(core->opt.cuda_max_readlen)/1000);

    return NULL;
}
    
pthread_t align_cudb_async(pthread_arg_t *pt_args,core_t* core, db_t* db, int32_t* ultra_long_reads, int32_t  n_ultra_long_reads) {
    assert(pt_args==NULL);
    pt_args = (pthread_arg_t *)malloc(sizeof(pthread_arg_t));
    MALLOC_CHK(pt_args);
    pt_args->core = core;
    pt_args->db = db;
    pt_args->starti = 0;
    pt_args->endi = n_ultra_long_reads;
    pt_args->ultra_long_reads=ultra_long_reads;

    pthread_t tid;    
    int ret = pthread_create(&tid, NULL, align_cudb,(void*)(pt_args));
    NEG_CHK(ret);

    return tid;
}

void align_cudb_async_join(pthread_arg_t *pt_args, pthread_t tid) {
    int ret = pthread_join(tid, NULL);
    NEG_CHK(ret);
    free(pt_args);


}

void align_cuda(core_t* core, db_t* db) {
    int32_t i,j;
    int32_t n_bam_rec = db->n_bam_rec;
    int32_t n_bam_rec_cuda;
    double realtime1;
    int32_t n_ultra_long_reads=0;
    int32_t ultra_long_reads[n_bam_rec];

    //cpu temp pointers
    int32_t* read_ptr_host;
    int32_t* n_events_host;
    int32_t* event_ptr_host;
    event_t* event_table_host;
    AlignedPair* event_align_pairs_host;
    int32_t* read_len_host;
    scalings_t* scalings_host;
    int32_t* n_event_align_pairs_host;
    char* read_host;

    /**cuda pointers*/
    char* read;        //flattened reads sequences
    int32_t* read_ptr; //index pointer for flattedned "reads"
    int32_t* read_len;
    int64_t sum_read_len;
    int32_t* n_events;
    event_t* event_table;
    int32_t* event_ptr;
    int64_t sum_n_events;
    scalings_t* scalings;
    AlignedPair* event_align_pairs;
    int32_t* n_event_align_pairs;
    half *bands;
    uint8_t *trace;
    EventKmerPair* band_lower_left;
    model_t* model_kmer_cache;
    model_t* model;

realtime1 = realtime();

    read_ptr_host = core->cuda->read_ptr_host;

    sum_read_len = 0;

    //read sequences : needflattening
    for (i = 0,j=0; i < n_bam_rec; i++) {
        if(db->read_len[i]<(core->opt.cuda_max_readlen)){
            read_ptr_host[j] = sum_read_len;
            sum_read_len += (db->read_len[i] + 1); //with null term
            j++;
        }
        else{
            ultra_long_reads[n_ultra_long_reads]=i;
            n_ultra_long_reads++;
        }
    }
    n_bam_rec_cuda = j;
    

    //can start processing on the ultra long reads on the CPU
    pthread_arg_t *tmparg=NULL;
    pthread_t tid =  align_cudb_async(tmparg,core, db, ultra_long_reads, n_ultra_long_reads);
    

    read_len_host = core->cuda->read_len_host;
    scalings_host = core->cuda->scalings_host;
    n_event_align_pairs_host = core->cuda->n_event_align_pairs_host;

    //form the temporary flattened array on host
    read_host = (char*)malloc(sizeof(char) * sum_read_len);
    MALLOC_CHK(read_host);
    for (i = 0,j=0; i < n_bam_rec; i++) {
        if(db->read_len[i]<(core->opt.cuda_max_readlen)){
            int32_t idx = read_ptr_host[j];
            strcpy(&read_host[idx], db->read[i]);
            read_len_host[j]=db->read_len[i];
            scalings_host[j]=db->scalings[i];
            j++;
        }

    }


    //now the events : need flattening
    //num events : need flattening
    //get the total size and create the pointers
    n_events_host = core->cuda->n_events_host;
    event_ptr_host = core->cuda->event_ptr_host;

    sum_n_events = 0;
    for (i = 0,j=0; i < n_bam_rec; i++) {
        if(db->read_len[i]<(core->opt.cuda_max_readlen)){
            n_events_host[j] = db->et[i].n;
            event_ptr_host[j] = sum_n_events;
            sum_n_events += db->et[i].n;
            j++;
        }
    }

    //event table flatten
    //form the temporary flattened array on host
    event_table_host =
        (event_t*)malloc(sizeof(event_t) * sum_n_events);
    MALLOC_CHK(event_table_host);
    for (i = 0,j=0; i < n_bam_rec; i++) {
        if(db->read_len[i]<(core->opt.cuda_max_readlen)){
            int32_t idx = event_ptr_host[j];
            memcpy(&event_table_host[idx], db->et[i].event,
                sizeof(event_t) * db->et[i].n);
                j++;
            }
    }

    event_align_pairs_host =
        (AlignedPair*)malloc(2 * sum_n_events * sizeof(AlignedPair));
    MALLOC_CHK(event_align_pairs_host);

core->align_cuda_preprocess += (realtime() - realtime1);

    /** Start GPU mallocs**/
realtime1 = realtime();


    read_ptr =core->cuda->read_ptr;
    read_len=core->cuda->read_len;
    n_events=core->cuda->n_events;
    event_ptr=core->cuda->event_ptr;
    scalings=core->cuda->scalings;
    model = core->cuda->model;


    print_size("read array",sum_read_len * sizeof(char));
    hipMalloc((void**)&read, sum_read_len * sizeof(char)); //with null char
    CUDA_CHK();
    print_size("event table",sum_n_events * sizeof(event_t));
    hipMalloc((void**)&event_table, sum_n_events * sizeof(event_t));
    CUDA_CHK();
    hipMalloc((void**)&model_kmer_cache, sum_read_len * sizeof(model_t)); 
    CUDA_CHK();
 
    /**allocate output arrays for cuda**/
    print_size("event align pairs",2 * sum_n_events *sizeof(AlignedPair));
    hipMalloc((void**)&event_align_pairs,
            2 * sum_n_events *
                sizeof(AlignedPair)); //todo : need better huristic
    CUDA_CHK();
    n_event_align_pairs=core->cuda->n_event_align_pairs;

    //scratch arrays
    size_t sum_n_bands = sum_n_events + sum_read_len; //todo : can be optimised 
    print_size("bands",sizeof(half) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&bands,sizeof(half) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    print_size("trace",sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&trace, sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    hipMemset(trace,0,sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH); //initialise the trace array to 0
    print_size("band_lower_left",sizeof(EventKmerPair)* sum_n_bands);
    hipMalloc((void**)&band_lower_left, sizeof(EventKmerPair)* sum_n_bands);
    CUDA_CHK();   
core->align_cuda_malloc += (realtime() - realtime1);

    /* cuda mem copys*/
realtime1 =realtime();
    hipMemcpy(read_ptr, read_ptr_host, n_bam_rec_cuda * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(read, read_host, sum_read_len * sizeof(char),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    //read length : already linear hence direct copy
    hipMemcpy(read_len, read_len_host, n_bam_rec_cuda * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(n_events, n_events_host, n_bam_rec_cuda * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(event_ptr, event_ptr_host, n_bam_rec_cuda * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    hipMemcpy(event_table, event_table_host, sizeof(event_t) * sum_n_events,
               hipMemcpyHostToDevice);
    CUDA_CHK();


    //can be interleaved
    hipMemcpy(scalings, scalings_host, sizeof(scalings_t) * n_bam_rec_cuda,
               hipMemcpyHostToDevice);
    CUDA_CHK();
core->align_cuda_memcpy += (realtime() - realtime1);



realtime1 = realtime();    
 
    /*pre kernel*/
    assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
    dim3 gridpre(1,(n_bam_rec_cuda + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
    dim3 blockpre(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);  
	fprintf(stderr,"grid %d,%d, block %d,%d\n",gridpre.x,gridpre.y, blockpre.x,blockpre.y);	

    align_kernel_pre_2d<<<gridpre, blockpre>>>( read,
        read_len, read_ptr, n_events,
        event_ptr, model, n_bam_rec_cuda, model_kmer_cache,bands,trace,band_lower_left); 
       
    hipDeviceSynchronize();CUDA_CHK();
    fprintf(stderr, "[%s::%.3f*%.2f] align pre done\n", __func__,
            realtime() - realtime1, cputime() / (realtime() - realtime1));
core->align_kernel_time += (realtime() - realtime1);        
core->align_pre_kernel_time += (realtime() - realtime1);        
                
realtime1 = realtime();

    /* core kernel*/
    assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
    dim3 grid1(1,(n_bam_rec_cuda + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
    dim3 block1(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);
    align_kernel_core_2d_shm<<<grid1, block1>>>(read_len, read_ptr, event_table, n_events,
            event_ptr, scalings, n_bam_rec_cuda, model_kmer_cache,bands,trace,band_lower_left );
    
    hipDeviceSynchronize();CUDA_CHK();
    fprintf(stderr, "[%s::%.3f*%.2f] align done\n", __func__,
    realtime() - realtime1, cputime() / (realtime() - realtime1));
    core->align_kernel_time += (realtime() - realtime1);
core->align_core_kernel_time += (realtime() - realtime1);

realtime1 = realtime();
    
    /*post kernel*/
    int32_t BLOCK_LEN = core->opt.cuda_block_size;
    dim3 gridpost((n_bam_rec_cuda + BLOCK_LEN - 1) / BLOCK_LEN);
    dim3 blockpost(BLOCK_LEN);
    #ifndef WARP_HACK  
        align_kernel_post<<<gridpost, blockpost>>>(event_align_pairs, n_event_align_pairs,
            read_len, read_ptr, event_table, n_events,
            event_ptr,scalings, n_bam_rec_cuda, model_kmer_cache,bands,trace,band_lower_left );     

    #else
        assert(BLOCK_LEN>=32);    
        dim3 grid1post((n_bam_rec_cuda + (BLOCK_LEN/32) - 1) / (BLOCK_LEN/32)); 
        fprintf(stderr,"grid new %d\n",grid1post.x);   
        align_kernel_post<<<grid1post, blockpost>>>(event_align_pairs, n_event_align_pairs,
            read_len, read_ptr, event_table, n_events,
            event_ptr, scalings, n_bam_rec_cuda, model_kmer_cache,bands,trace,band_lower_left );
    #endif
    hipDeviceSynchronize();CUDA_CHK();
    fprintf(stderr, "[%s::%.3f*%.2f] align post done\n", __func__,
            realtime() - realtime1, cputime() / (realtime() - realtime1));
    core->align_kernel_time += (realtime() - realtime1);        
core->align_post_kernel_time += (realtime() - realtime1);        


    //fprintf(stderr,"readlen %d,n_events %d\n",db->read_len[i],n_event_align_pairs);

#ifdef CUDA_DEBUG

    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    //todo : print a message to detect the launch timed out
    if (code == hipErrorLaunchTimeOut) {
        ERROR("%s", "The kernel timed out. You have to first disable the cuda "
                    "time out.");
        fprintf(
            stderr,
            "On Ubuntu do the following\nOpen the file /etc/X11/xorg.conf\nYou "
            "will have a section about your NVIDIA device. Add the following "
            "line to it.\nOption \"Interactive\" \"0\"\nIf you do not have a "
            "section about your NVIDIA device in /etc/X11/xorg.conf or you do "
            "not have a file named /etc/X11/xorg.conf, run the command sudo "
            "nvidia-xconfig to generate a xorg.conf file and do as above.\n\n");
    }
    if (code != hipSuccess) {
        fprintf(stderr, "Cuda error: %s \n in file : %s line number : %lu\n",
                hipGetErrorString(code), __FILE__, __LINE__);
        exit(-1);
    }        
    
#endif

    /** copyback ans**/
realtime1 =  realtime();
    hipMemcpy(n_event_align_pairs_host, n_event_align_pairs,
               n_bam_rec_cuda * sizeof(int32_t), hipMemcpyDeviceToHost);
    CUDA_CHK();
     
    hipMemcpy(event_align_pairs_host, event_align_pairs,
               2 * sum_n_events * sizeof(AlignedPair), hipMemcpyDeviceToHost);
    CUDA_CHK();
core->align_cuda_memcpy += (realtime() - realtime1);

realtime1 =  realtime();

    hipFree(read); //with null char
    hipFree(event_table);
    hipFree(event_align_pairs);
    hipFree(bands);
    hipFree(trace);
    hipFree(band_lower_left);
    hipFree(model_kmer_cache);

core->align_cuda_malloc += (realtime() - realtime1);    
    
    /** post work**/
realtime1 =  realtime();
    //copy back
    for (i = 0,j=0; i < n_bam_rec; i++) {
        if(db->read_len[i]<(core->opt.cuda_max_readlen)){
            int32_t idx = event_ptr_host[j];
            db->n_event_align_pairs[i]=n_event_align_pairs_host[j];
            memcpy(db->event_align_pairs[i], &event_align_pairs_host[idx * 2],
                sizeof(AlignedPair) * db->n_event_align_pairs[i]);
                j++;
        }
    }

    //free the temp arrays on host
    free(read_host);
    free(event_table_host);
    free(event_align_pairs_host);

core->align_cuda_postprocess += (realtime() - realtime1);

realtime1 =  realtime(); 
    align_cudb_async_join(tmparg,tid);  
core->extra_load_cpu += (realtime() - realtime1);


}


#endif
